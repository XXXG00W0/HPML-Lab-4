#include "hip/hip_runtime.h"
// conv.cu

#include "convolutionKernel.h"

__global__ void basicConvolution2d(const InputTensor in_tensor, const FilterTensor filter, OutputTensor out_tensor){
    
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;
    int out_k = blockIdx.z;

    if (out_x >= out_tensor.W || out_y >= out_tensor.H) return;

    double conv_sum = 0.0;

    for (int c = 0; c < in_tensor.C; ++c){
        for (int j = 0; j < filter.FH; ++j){
            for (int i = 0; i < filter.FW; ++i){

                // calcualte the position of the filter
                int filter_pos = out_k * filter.C * filter.FW * filter.FH 
                            + c * filter.FW * filter.FH
                            + j * filter.FW + i;

                // calculate the position of the input tensor
                int in_x = out_x + i - filter.FW/2;
                int in_y = out_y + j - filter.FH/2;
                
                // check if the position is valid
                if (in_x >= 0 && in_x < in_tensor.W && in_y > 0 && in_y < in_tensor.H){
                    int in_pos = c * in_tensor.H * in_tensor.W + in_y * in_tensor.W + in_x;
                    conv_sum += filter.elements[filter_pos] * in_tensor.elements[in_pos];
                } 
            }
        }
    }
    
    int out_pos = out_k * out_tensor.H * out_tensor.W + out_x * out_tensor.W + out_y;
    out_tensor.elements[out_pos] = conv_sum;
    
}

__global__ void tiledConvolution2d(const InputTensor in_tensor, const FilterTensor filter, OutputTensor out_tensor){
    float *in_sub, *out_sub;

    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;
    int out_k = blockIdx.z;

    int shared_y = threadIdx.y + filter.FH/2;
    int shared_x = threadIdx.x + filter.FW/2;

    __shared__ double in_shared[BLOCK_SIZE+2][BLOCK_SIZE+2];
    __shared__ double filter_shared[BLOCK_SIZE][BLOCK_SIZE];

    // load input tensor to shared memory
    // assumed that in_tensor is padded
    if (out_x < in_tensor.W && out_y < in_tensor.H){
        in_shared[shared_x][shared_y] = in_tensor.elements[out_x * in_tensor.W + out_y];
    }

    // load filter tensor to shared memory
    if (shared_x < filter.FW && shared_y < filter.FH){
        filter_shared[shared_x][shared_y] = filter.elements[out_k * filter.C * filter.FW * filter.FH + shared_x * filter.FW + shared_y];
    }

    __syncthreads();

    // calculate the convolution
    double conv_sum = 0.0;
    if (out_x < out_tensor.W && out_y < out_tensor.H){
        for (int c = 0; c < in_tensor.C; ++c){
            for (int j = 0; j < in_tensor.H; ++j){
                for (int i = 0; i < in_tensor.W; ++i){
                    conv_sum += filter_shared[j][i] * in_shared[shared_x + i][shared_y + j];
                }
            }
        }
    }

    // store the result
    int out_pos = out_k * out_tensor.H * out_tensor.W + out_x * out_tensor.W + out_y;
    out_tensor.elements[out_pos] = conv_sum;
}

__host__ InputTensor tensorPadding(const InputTensor in_tensor){
    InputTensor padded;
    padded.C = in_tensor.C;
    padded.H = in_tensor.H + 2;
    padded.W = in_tensor.W + 2;
    size_t size = padded.C * padded.H * padded.W * sizeof(double);
    padded.elements = (double*)malloc(size);
    for (int c = 0; c < padded.C; c++){
        for (int h = 0; h < padded.H; h++){
            for (int w = 0; w < padded.W; w++){
                if (h == 0 || h == padded.H - 1 || w == 0 || w == padded.W - 1){
                    padded.elements[c * padded.H * padded.W + h * padded.W + w] = 0;
                } else {
                    padded.elements[c * padded.H * padded.W + h * padded.W + w] = in_tensor.elements[c * in_tensor.H * in_tensor.W + (h - 1) * in_tensor.W + (w - 1)];
                }
            }
        }
    }
    return padded;
}

__host__ InputTensor buildHostInputTensor(int C, int H, int W){

    InputTensor in_tensor;
    in_tensor.C = C;
    in_tensor.H = H;
    in_tensor.W = W;
    size_t size = C * H * W * sizeof(double);
    in_tensor.elements = (double*)malloc(size);
    return in_tensor;
}

__host__ InputTensor buildDeviceInputTensor(InputTensor in_tensor, bool copy){
    
    InputTensor in_tensor_d;
    in_tensor_d.C = in_tensor.C;
    in_tensor_d.H = in_tensor.H;
    in_tensor_d.W = in_tensor.W;
    size_t size = in_tensor.C * in_tensor.H * in_tensor.W * sizeof(double);
    hipMalloc((void**)&in_tensor_d.elements, size);
    if(copy){
        hipMemcpy(in_tensor_d.elements, in_tensor.elements, size, hipMemcpyHostToDevice);
    }
    return in_tensor_d;
}

__host__ FilterTensor buildHostFilterTensor(int K, int C, int FH, int FW){

    FilterTensor f_tensor;
    f_tensor.K = K;
    f_tensor.C = C;
    f_tensor.FH = FH;
    f_tensor.FW = FW;
    size_t size = K * C * FH * FW * sizeof(double);
    f_tensor.elements = (double*) malloc(size);
    return f_tensor;
}

__host__ FilterTensor buildDeviceFilterTensor(FilterTensor f_tensor, bool copy){
    
    FilterTensor f_tensor_d;
    f_tensor_d.K = f_tensor.K;
    f_tensor_d.C = f_tensor.C;
    f_tensor_d.FH = f_tensor.FH;
    f_tensor_d.FW = f_tensor.FW;
    size_t size = f_tensor.K * f_tensor.C * f_tensor.FH * f_tensor.FW * sizeof(double);
    hipMalloc((void**)&f_tensor_d.elements, size);
    if(copy){
        hipMemcpy(f_tensor_d.elements, f_tensor.elements, size, hipMemcpyHostToDevice);
    }
    return f_tensor_d;
}

__host__ OutputTensor buildHostOutputTensor(int K, int H, int W){

    OutputTensor out_tensor;
    out_tensor.K = K;
    out_tensor.H = H;
    out_tensor.W = W;
    size_t size = K * H * W * sizeof(double);
    out_tensor.elements = (double*)malloc(size);
    return out_tensor;
}

__host__ OutputTensor buildDeviceOutputTensor(OutputTensor out_tensor, bool copy){
    
    OutputTensor out_tensor_d;
    out_tensor_d.K = out_tensor.K;
    out_tensor_d.H = out_tensor.H;
    out_tensor_d.W = out_tensor.W;
    size_t size = out_tensor.K * out_tensor.H * out_tensor.W * sizeof(double);
    hipMalloc((void**)&out_tensor_d.elements, size);
    if(copy){
        hipMemcpy(out_tensor_d.elements, out_tensor.elements, size, hipMemcpyHostToDevice);
    }
    return out_tensor_d;
}

__host__ void initInputTensor(InputTensor in_tensor){
    // size_t size = in_tensor.C * in_tensor.H * in_tensor.W * sizeof(double);
    double value;

    for (int c = 0; c < in_tensor.C; c++){
        for (int h = 0; h < in_tensor.H; h++){
            for (int w = 0; w < in_tensor.W; w++){
                value =  c * (h + w);
                in_tensor.elements[c * in_tensor.H * in_tensor.W + h * in_tensor.W + w] = value;
            }
        }
    }
}

__host__ void initFilterTensor(FilterTensor f_tensor){
    // size_t size = f_tensor.K * f_tensor.C * f_tensor.FW * f_tensor.FH;
    double value;

    for (int k = 0; k < f_tensor.K; k++){
        for (int c = 0; c < f_tensor.C; c++){
            for (int fw = 0; fw < f_tensor.FW; fw++){
                for (int fh = 0; fh < f_tensor.FH; fh++){
                    value = (c + k) * (fw + fh);
                    f_tensor.elements[k * f_tensor.C * f_tensor.FW * f_tensor.FH + c * 
                    f_tensor.FW * f_tensor.FH + fw * f_tensor.FH + fh] = value;
                }
            }
        }
    }
}

__host__ double checksum(OutputTensor out_tensor){
    double sum = 0.0;
    for (int k = 0; k < out_tensor.K; k++){
        for (int h = 0; h < out_tensor.H; h++){
            for (int w = 0; w < out_tensor.W; w++){
                sum += out_tensor.elements[k * out_tensor.H * out_tensor.W + h * out_tensor.W + w];
            }
        }
    }
    return sum;
}